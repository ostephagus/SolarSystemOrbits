#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif
    float* __declspec(dllexport) challengeOne();//non cuda
#ifdef __cplusplus
}
#endif // __cplusplus

#ifdef __cplusplus
extern "C" {
#endif
    float* __declspec(dllexport) challengeTwo();// non cuda
#ifdef __cplusplus
}
#endif // __cplusplus

#ifdef __cplusplus
extern "C" {
#endif
    float* __declspec(dllexport) challengeThree(); //
#ifdef __cplusplus
}
#endif // __cplusplus

#ifdef __cplusplus
extern "C" {
#endif
    float* __declspec(dllexport) challengeFour(); //can be cuda as rotation matrix
#ifdef __cplusplus
}
#endif // __cplusplus

#ifdef __cplusplus
extern "C" {
#endif
    float* __declspec(dllexport) challengeFive(); //can be cuda as lots of samples
#ifdef __cplusplus
}
#endif // __cplusplus

#ifdef __cplusplus
extern "C" {
#endif
    float* __declspec(dllexport) challengeSix(); //can be cuda as image
#ifdef __cplusplus
}
#endif // __cplusplus

#ifdef __cplusplus
extern "C" {
#endif
    float* __declspec(dllexport) challengeSeven(); //can be cuda as image data
#ifdef __cplusplus
}
#endif // __cplusplus

float* challengeOne() {

}

