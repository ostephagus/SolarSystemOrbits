#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>
using namespace std;
extern "C" {
    void __declspec(dllexport) challengeOne();//non cuda
}
extern "C" {
void __declspec(dllexport) challengeTwo();// non cuda
}
extern "C" {
void __declspec(dllexport) challengeThree(); //
}
extern "C" {
void __declspec(dllexport) challengeFour(); //can be cuda as rotation matrix
}
extern "C" {
void __declspec(dllexport) challengeFive(); //can be cuda as lots of samples
}
extern "C" {
void __declspec(dllexport) challengeSix(); //can be cuda as image
}
extern "C" {
void __declspec(dllexport) challengeSeven(); //can be cuda as image data
}
float radius(float semimaj, float e, float theta) {
    return (semimaj * (1 - powf(e, 2))) / (1 - cosf(theta));
}

void challengeOne(float *data) {
    float radii[8] = { 0.387, 0.723, 1, 1.523, 5.2, 9.58, 19.29, 30.25 };
    float periods[8] = { 0.24, 0.62, 1, 1.88, 11.86, 29.63, 84.75, 166.34 };
    float mean = 0;
    float r2;
    for (int i = 0; i < 8; i++) {
        mean += periods[i] / 8.0f;
    }
    float rss=0;
    float css = 0;
    for (int i = 0; i < 8; i++) {
        rss += powf(powf(periods[i]- sqrt(radii[i]), 3), 2);
        css += powf(periods[i] - mean, 2);
    }

    for (int i = 0; i < 8; i++) {
        data[i] = radii[i];
    }
    for (int i = 8; i < 16; i++) {
        data[i] = periods[i%8];
    }
    r2 = 1 - (rss / css);
    data[16] = r2;
}

void challengeTwo(float* d_out, float semimaj, float e) {
    float dtheta = 0.002 * M_PI;
    //float ttheta = 2 * M_PI;
    
    float theta = 0;
    float rad;
    for (int i = 0; i < 2000; i++) {
        theta += dtheta;
        rad = radius(semimaj, e, theta);
        d_out[i] = rad * cosf(theta);
        d_out[i+1000] = rad * sinf(theta);
    }
}

void challengeThree(float* data) {

}