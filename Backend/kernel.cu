#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>;
using namespace std;

float* __declspec(dllexport) challengeOne();//non cuda

float* __declspec(dllexport) challengeTwo();// non cuda

float* __declspec(dllexport) challengeThree(); //

float* __declspec(dllexport) challengeFour(); //can be cuda as rotation matrix

float* __declspec(dllexport) challengeFive(); //can be cuda as lots of samples

float* __declspec(dllexport) challengeSix(); //can be cuda as image

float* __declspec(dllexport) challengeSeven(); //can be cuda as image data

float radius(float semimaj, float e, float theta) {
    return (semimaj * (1 - powf(e, 2))) / (1 - cosf(theta));
}

float* challengeOne() {
    float radii[8] = { 0.387, 0.723, 1, 1.523, 5.2, 9.58, 19.29, 30.25 };
    float periods[8] = { 0.24, 0.62, 1, 1.88, 11.86, 29.63, 84.75, 166.34 };
    float mean = 0;
    float r2;
    for (int i = 0; i < 8; i++) {
        mean += periods[i] / 8.0f;
    }
    float rss=0;
    float css = 0;
    for (int i = 0; i < 8; i++) {
        rss += powf(powf(periods[i]- sqrt(radii[i]), 3), 2);
        css += powf(periods[i] - mean, 2);
    }

    float* d_out = (float*)malloc(sizeof(float) * 17);
    for (int i = 0; i < 8; i++) {
        d_out[i] = radii[i];
    }
    for (int i = 8; i < 16; i++) {
        d_out[i] = periods[i%8];
    }
    r2 = 1 - (rss / css);
    d_out[16] = r2;
    return d_out;
}

float* challengeTwo(float semimaj, float e) {
    float dtheta = 0.002 * M_PI;
    float ttheta = 2 * M_PI;

    
    float theta = 0;
    float rad;
    float* dout = (float*)malloc(4000 * sizeof(float));
    for (int i = 0; i < 2000; i++) {
        theta += dtheta;
        rad = radius(semimaj, e, theta);
        dout[i] = rad * cosf(theta);
        dout[i+1000] = rad * sinf(theta);
    }
    return dout;

}